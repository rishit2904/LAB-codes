#include <stdio.h>
#include <hip/hip_runtime.h>
#define N 10  

__global__ void oddKernel(int *arr, int n) {
    int i = 2 * threadIdx.x + 1;
    if (i < n - 1 && arr[i] > arr[i + 1]) {
        int temp = arr[i];
        arr[i] = arr[i + 1];
        arr[i + 1] = temp;
    }
}

__global__ void evenKernel(int *arr, int n) {
    int i = 2 * threadIdx.x;
    if (i < n - 1 && arr[i] > arr[i + 1]) {
        int temp = arr[i];
        arr[i] = arr[i + 1];
        arr[i + 1] = temp;
    }
}

int main() {
    int h_arr[N];
    int *d_arr;

    printf("Enter %d elements: ", N);
    for (int i = 0; i < N; i++) {
        scanf("%d", &h_arr[i]);
    }

    hipMalloc((void **)&d_arr, N * sizeof(int));
    hipMemcpy(d_arr, h_arr, N * sizeof(int), hipMemcpyHostToDevice);

    int threadsPerBlock = N / 2;
    for (int i = 0; i < N; i++) {
        oddKernel<<<1, threadsPerBlock>>>(d_arr, N);
        hipDeviceSynchronize();
        evenKernel<<<1, threadsPerBlock>>>(d_arr, N);
        hipDeviceSynchronize();
    }

    hipMemcpy(h_arr, d_arr, N * sizeof(int), hipMemcpyDeviceToHost);

    printf("Sorted array: ");
    for (int i = 0; i < N; i++) {
        printf("%d ", h_arr[i]);
    }
    printf("\n");

    hipFree(d_arr);
    return 0;
}
