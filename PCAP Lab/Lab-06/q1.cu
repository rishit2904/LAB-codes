#include <stdio.h>
#include <hip/hip_runtime.h>

#define MASK_WIDTH 3

// CUDA Kernel for 1D Convolution
__global__ void convolution_1D(float *N, float *M, float *P, int width) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    float sum = 0.0f;

    if (i < width) {
        for (int j = 0; j < MASK_WIDTH; j++) {
            int index = i - MASK_WIDTH / 2 + j;
            if (index >= 0 && index < width) {
                sum += N[index] * M[j];
            }
        }
        P[i] = sum;
    }
}

int main() {
    const int width = 10;
    float h_N[width] = {1, 2, 3, 4, 5, 6, 7, 8, 9, 10};
    float h_M[MASK_WIDTH] = {0.2, 0.5, 0.2};
    float h_P[width];

    float *d_N, *d_M, *d_P;

    hipMalloc((void **)&d_N, width * sizeof(float));
    hipMalloc((void **)&d_M, MASK_WIDTH * sizeof(float));
    hipMalloc((void **)&d_P, width * sizeof(float));

    hipMemcpy(d_N, h_N, width * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_M, h_M, MASK_WIDTH * sizeof(float), hipMemcpyHostToDevice);

    int blockSize = 256;
    int gridSize = (width + blockSize - 1) / blockSize;
    convolution_1D<<<gridSize, blockSize>>>(d_N, d_M, d_P, width);

    hipMemcpy(h_P, d_P, width * sizeof(float), hipMemcpyDeviceToHost);

    printf("Output array P:\n");
    for (int i = 0; i < width; i++) {
        printf("%f ", h_P[i]);
    }
    printf("\n");

    hipFree(d_N);
    hipFree(d_M);
    hipFree(d_P);

    return 0;
}