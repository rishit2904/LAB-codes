#include <stdio.h>
#include <hip/hip_runtime.h>
#define N 10  

__global__ void selectionSort(int *arr, int n) {
    for (int i = 0; i < n - 1; i++) {
        int minIndex = i;
        for (int j = i + 1; j < n; j++) {
            if (arr[j] < arr[minIndex]) {
                minIndex = j;
            }
        }
        __syncthreads();

        if (minIndex != i) {
            int temp = arr[i];
            arr[i] = arr[minIndex];
            arr[minIndex] = temp;
        }
        __syncthreads();
    }
}

int main() {
    int h_arr[N];
    int *d_arr;

    printf("Enter %d elements: ", N);
    for (int i = 0; i < N; i++) {
        scanf("%d", &h_arr[i]);
    }

    hipMalloc((void **)&d_arr, N * sizeof(int));
    hipMemcpy(d_arr, h_arr, N * sizeof(int), hipMemcpyHostToDevice);

    selectionSort<<<1, 1>>>(d_arr, N);
    hipMemcpy(h_arr, d_arr, N * sizeof(int), hipMemcpyDeviceToHost);

    printf("Sorted array: ");
    for (int i = 0; i < N; i++) {
        printf("%d ", h_arr[i]);
    }
    printf("\n");

    hipFree(d_arr);
    return 0;
}
