#include<hip/hip_runtime.h>

#include <stdio.h>
#include <string.h>

__global__ void appendStringAtomic(char *output, char *input, int inputLength, int *pos) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < inputLength) {

        int sublen=inputLength-idx;
        int p=atomicAdd(pos,sublen);
        for (int i = 0; i <sublen; i++)
        {
            output[p+i]=input[i];
        }
    }
}

int main() {
    char *input = "PCAP";
    int inputLength = strlen(input);
    int outputLength = inputLength * (inputLength+1)/2;
    printf("Input String: %s\n",input);

    char *d_input, *d_output;
    int *d_pos, *d_subsize;

    hipEvent_t start,stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start,0);
    hipMalloc(&d_input, inputLength * sizeof(char));
    hipMalloc(&d_output, outputLength * sizeof(char));  
    hipMalloc(&d_pos, sizeof(int));  
    hipMalloc(&d_subsize, sizeof(int));

    hipMemcpy(d_input, input, inputLength * sizeof(char), hipMemcpyHostToDevice);

    hipError_t error=hipGetLastError();
    if(error!=hipSuccess)
    printf("CUDA Error 1: %s\n",hipGetErrorString(error));

    hipMemset(d_output, 0, outputLength * sizeof(char)); 
    hipMemset(d_pos, 0, sizeof(int));  

    appendStringAtomic<<<1, 256>>>(d_output, d_input, inputLength, d_pos);
    

    error=hipGetLastError();
    if(error!=hipSuccess)
    printf("CUDA Error 2: %s\n",hipGetErrorString(error));

    hipEventRecord(stop,0);
    hipEventSynchronize(stop);
    float elapsed;
    hipEventElapsedTime(&elapsed,start,stop);

    char *h_output = new char[outputLength + 1];
    hipMemcpy(h_output, d_output, outputLength * sizeof(char), hipMemcpyDeviceToHost);  // Copy output string

    h_output[outputLength] = '\0';  // Null-terminate the string

    printf("Result: %s\n",h_output);
    printf("Time taken = %f\n",elapsed);

    // Clean up
    free(h_output);
    hipFree(d_input);
    hipFree(d_output);
    hipFree(d_subsize);
    hipFree(d_pos);
    
    return 0;
}
