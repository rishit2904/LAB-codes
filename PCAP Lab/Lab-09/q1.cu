#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>


__global__ void spmv_csr_kernel(
    int num_rows,
    const float *csrVal,
    const int *csrRowPtr,
    const int *csrColInd,
    const float *x,
    float *y)
{
    int row = blockIdx.x * blockDim.x + threadIdx.x;  
    if (row < num_rows) {
        float sum = 0.0f;
        int row_start = csrRowPtr[row];
        int row_end = csrRowPtr[row + 1];
        for (int i = row_start; i < row_end; i++) {
            sum += csrVal[i] * x[csrColInd[i]];
        }
        y[row] = sum;
    }
}
void spmv_csr(
    int num_rows,
    int num_cols,
    int nnz,              
    float *csrVal,        
    int *csrRowPtr,       
    int *csrColInd,       
    float *x,             
    float *y)             
{

    float *d_csrVal, *d_x, *d_y;
    int *d_csrRowPtr, *d_csrColInd;
    hipMalloc(&d_csrVal, nnz * sizeof(float));
    hipMalloc(&d_csrRowPtr, (num_rows + 1) * sizeof(int));
    hipMalloc(&d_csrColInd, nnz * sizeof(int));
    hipMalloc(&d_x, num_cols * sizeof(float));
    hipMalloc(&d_y, num_rows * sizeof(float));
    hipMemcpy(d_csrVal, csrVal, nnz * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_csrRowPtr, csrRowPtr, (num_rows + 1) * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_csrColInd, csrColInd, nnz * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_x, x, num_cols * sizeof(float), hipMemcpyHostToDevice);
    int threadsPerBlock = 256;
    int blocksPerGrid = (num_rows + threadsPerBlock - 1) / threadsPerBlock;
    spmv_csr_kernel<<<blocksPerGrid, threadsPerBlock>>>(
        num_rows, d_csrVal, d_csrRowPtr, d_csrColInd, d_x, d_y);


    hipMemcpy(y, d_y, num_rows * sizeof(float), hipMemcpyDeviceToHost);
    hipFree(d_csrVal);
    hipFree(d_csrRowPtr);
    hipFree(d_csrColInd);
    hipFree(d_x);
    hipFree(d_y);
}

int main() { 
    int num_rows = 4;
    int num_cols = 4;
    int nnz = 7;  
    float csrVal[] = {5, 2, 3, 1, 4, 6};           
    int csrRowPtr[] = {0, 2, 3, 5, 6, 7};         
    int csrColInd[] = {0, 2, 1, 0, 3, 2};          
    float x[] = {1, 2, 3, 4};
    float *y = (float*)malloc(num_rows * sizeof(float));
    spmv_csr(num_rows, num_cols, nnz, csrVal, csrRowPtr, csrColInd, x, y);
    printf("Result vector y:\n");
    for (int i = 0; i < num_rows; i++) {
        printf("y[%d] = %.2f\n", i, y[i]);
    }

    free(y);
    return 0;
}
