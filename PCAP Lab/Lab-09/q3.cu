#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

// CUDA kernel to create matrix B with 1's complement for non-border elements
__global__ void onesComplementNonBorder(float *d_A, float *d_B, int M, int N) {
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;

    if (row < M && col < N) {
        int idx = row * N + col;
        float value = d_A[idx];

        // Check if element is non-border
        bool isBorder = (row == 0 || row == M-1 || col == 0 || col == N-1);

        if (!isBorder) {
            // Apply 1's complement (assuming binary values: 0->1, 1->0)
            int val= value;
            int binVal = 0;
            int size = 0;
            while(val>0)
            {
                int rem = val%2==0?1:0;
                binVal = rem*(int)(pow(10, size)) + binVal;
                size++;
                val/=2;
            }
            d_B[idx] = binVal;
        } else {
            // Keep border elements unchanged
            d_B[idx] = value;
        }
    }
}

// Function to print matrix
void printMatrix(float *matrix, int M, int N) {
    for (int i = 0; i < M; i++) {
        for (int j = 0; j < N; j++) {
            printf("%.0f\t", matrix[i * N + j]);
        }
        printf("\n");
    }
    printf("\n");
}

int main() {
    // Matrix dimensions
    int M = 4;  // rows
    int N = 4;  // columns

    // Host matrices
    float *h_A = (float*)malloc(M * N * sizeof(float));
    float *h_B = (float*)malloc(M * N * sizeof(float));

    // Initialize matrix A with sample binary values
    float values[M][N] = {
        {1, 2, 3, 4},
        {6, 5, 8, 3},
        {2, 4, 10, 1},
        {9, 1, 2, 5}
    };

    // Copy values to 1D array
    for (int i = 0; i < M; i++) {
        for (int j = 0; j < N; j++) {
            h_A[i * N + j] = values[i][j];
        }
    }

    printf("Input Matrix A:\n");
    printMatrix(h_A, M, N);

    // Device matrices
    float *d_A, *d_B;
    hipMalloc(&d_A, M * N * sizeof(float));
    hipMalloc(&d_B, M * N * sizeof(float));

    // Copy input matrix to device
    hipMemcpy(d_A, h_A, M * N * sizeof(float), hipMemcpyHostToDevice);

    // Set up execution configuration
    dim3 threadsPerBlock(16, 16);
    dim3 blocksPerGrid(
        (N + threadsPerBlock.x - 1) / threadsPerBlock.x,
        (M + threadsPerBlock.y - 1) / threadsPerBlock.y
    );

    // Launch kernel
    onesComplementNonBorder<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, M, N);

    // Copy result back to host
    hipMemcpy(h_B, d_B, M * N * sizeof(float), hipMemcpyDeviceToHost);

    // Print output matrix
    printf("Output Matrix B:\n");
    printMatrix(h_B, M, N);

    // Free memory
    hipFree(d_A);
    hipFree(d_B);
    free(h_A);
    free(h_B);

    return 0;
}
