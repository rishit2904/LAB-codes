#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>


__global__ void modifyMatrixRows(float *d_A, int M, int N) {
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;

    if (row < M && col < N) {
        int idx = row * N + col;
        float value = d_A[idx];


        if (row > 0) {
            float result = value;
            for (int p = 1; p < row + 1; p++) {
                result *= value;
            }
            d_A[idx] = result;
        }
    }
}


void printMatrix(float *matrix, int M, int N) {
    for (int i = 0; i < M; i++) {
        for (int j = 0; j < N; j++) {
            printf("%.2f\t", matrix[i * N + j]);
        }
        printf("\n");
    }
    printf("\n");
}

int main() {

    int M = 4;  
    int N = 3;  


    float *h_A = (float*)malloc(M * N * sizeof(float));


    float values[M][N] = {
        {1.0, 2.0, 3.0},
        {1.0, 2.0, 3.0},
        {1.0, 2.0, 3.0},
        {1.0, 2.0, 3.0}
    };


    for (int i = 0; i < M; i++) {
        for (int j = 0; j < N; j++) {
            h_A[i * N + j] = values[i][j];
        }
    }

    printf("Original Matrix:\n");
    printMatrix(h_A, M, N);


    float *d_A;
    hipMalloc(&d_A, M * N * sizeof(float));


    hipMemcpy(d_A, h_A, M * N * sizeof(float), hipMemcpyHostToDevice);


    dim3 threadsPerBlock(16, 16);
    dim3 blocksPerGrid(
        (N + threadsPerBlock.x - 1) / threadsPerBlock.x,
        (M + threadsPerBlock.y - 1) / threadsPerBlock.y
    );


    modifyMatrixRows<<<blocksPerGrid, threadsPerBlock>>>(d_A, M, N);


    hipMemcpy(h_A, d_A, M * N * sizeof(float), hipMemcpyDeviceToHost);


    printf("Modified Matrix:\n");
    printMatrix(h_A, M, N);


    hipFree(d_A);
    free(h_A);

    return 0;
}
