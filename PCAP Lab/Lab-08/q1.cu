#include <stdio.h>
#include <hip/hip_runtime.h>

#define N 3  // Matrix size

// Kernel: Each row computed by one thread
__global__ void matrixAddRow(int *A, int *B, int *C, int n) {
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    if (row < n) {
        for (int col = 0; col < n; col++) {
            C[row * n + col] = A[row * n + col] + B[row * n + col];
        }
    }
}

// Kernel: Each column computed by one thread
__global__ void matrixAddCol(int *A, int *B, int *C, int n) {
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    if (col < n) {
        for (int row = 0; row < n; row++) {
            C[row * n + col] = A[row * n + col] + B[row * n + col];
        }
    }
}

// Kernel: Each element computed by one thread
__global__ void matrixAddElement(int *A, int *B, int *C, int n) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    if (row < n && col < n) {
        C[row * n + col] = A[row * n + col] + B[row * n + col];
    }
}

// Function to print matrix
void printMatrix(const char* msg, int C[N][N]) {
    printf("%s\n", msg);
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++)
            printf("%d ", C[i][j]);
        printf("\n");
    }
    printf("\n");
}

int main() {
    int size = N * N * sizeof(int);
    int h_A[N][N], h_B[N][N], h_C[N][N];

    // Taking user input
    printf("Enter elements of Matrix A (%dx%d):\n", N, N);
    for (int i = 0; i < N; i++)
        for (int j = 0; j < N; j++)
            scanf("%d", &h_A[i][j]);

    printf("Enter elements of Matrix B (%dx%d):\n", N, N);
    for (int i = 0; i < N; i++)
        for (int j = 0; j < N; j++)
            scanf("%d", &h_B[i][j]);

    int *d_A, *d_B, *d_C_row, *d_C_col, *d_C_element;

    // Allocate memory on GPU
    hipMalloc((void**)&d_A, size);
    hipMalloc((void**)&d_B, size);
    hipMalloc((void**)&d_C_row, size);
    hipMalloc((void**)&d_C_col, size);
    hipMalloc((void**)&d_C_element, size);

    // Copy input matrices to GPU
    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    // 1. Row-wise computation
    matrixAddRow<<<N, 1>>>(d_A, d_B, d_C_row, N);
    hipMemcpy(h_C, d_C_row, size, hipMemcpyDeviceToHost);
    printMatrix("Matrix Addition (Row-wise Computation):", h_C);

    // 2. Column-wise computation
    matrixAddCol<<<N, 1>>>(d_A, d_B, d_C_col, N);
    hipMemcpy(h_C, d_C_col, size, hipMemcpyDeviceToHost);
    printMatrix("Matrix Addition (Column-wise Computation):", h_C);

    // 3. Element-wise computation
    dim3 threadsPerBlock(16, 16);
    dim3 blocksPerGrid((N + 15) / 16, (N + 15) / 16);
    matrixAddElement<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C_element, N);
    hipMemcpy(h_C, d_C_element, size, hipMemcpyDeviceToHost);
    printMatrix("Matrix Addition (Element-wise Computation):", h_C);

    // Free memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C_row);
    hipFree(d_C_col);
    hipFree(d_C_element);

    return 0;
}
