#include <iostream>
#include <hip/hip_runtime.h>

#define N 1024   // Length of the input signal
#define K 5      // Length of the kernel

// Declare constant memory for the kernel
__constant__ float d_kernel[K];

// Kernel for 1D convolution using constant memory
__global__ void convolution1D(float* d_input, float* d_output, int signal_size, int kernel_size) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index < signal_size) {
        float result = 0.0f;
        // Perform convolution by iterating over the kernel
        for (int i = 0; i < kernel_size; i++) {
            // Handle the boundary conditions
            if (index - i >= 0)
                result += d_input[index - i] * d_kernel[i];
        }
        d_output[index] = result;
    }
}

int main() {
    // Host arrays
    float h_input[N], h_kernel[K], h_output[N];
    float *d_input, *d_output;

    // Initialize the input signal and kernel with some values
    for (int i = 0; i < N; i++) {
        h_input[i] = rand() % 100;
    }
    for (int i = 0; i < K; i++) {
        h_kernel[i] = rand() % 10;
    }

    // Allocate device memory
    hipMalloc(&d_input, N * sizeof(float));
    hipMalloc(&d_output, N * sizeof(float));

    // Copy input and kernel data to the device
    hipMemcpy(d_input, h_input, N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(d_kernel), h_kernel, K * sizeof(float));

    // Define grid and block size
    int blockSize = 256;
    int gridSize = (N + blockSize - 1) / blockSize;

    // Launch the kernel
    convolution1D<<<gridSize, blockSize>>>(d_input, d_output, N, K);

    // Copy the result back to the host
    hipMemcpy(h_output, d_output, N * sizeof(float), hipMemcpyDeviceToHost);

    // Print the output (first 10 values for brevity)
    std::cout << "Convolution Output (first 10 values):" << std::endl;
    for (int i = 0; i < 10; i++) {
        std::cout << h_output[i] << " ";
    }
    std::cout << std::endl;

    // Free memory
    hipFree(d_input);
    hipFree(d_output);

    return 0;
}
