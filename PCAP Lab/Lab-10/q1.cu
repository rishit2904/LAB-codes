#include <iostream>
#include <hip/hip_runtime.h>

#define N 16  // Size of the matrix

// Kernel to perform matrix multiplication
__global__ void matrixMulCUDA(int* A, int* B, int* C, int width) {
    // Get the row and column index of the element to compute
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < width && col < width) {
        int value = 0;
        for (int k = 0; k < width; k++) {
            value += A[row * width + k] * B[k * width + col];
        }
        C[row * width + col] = value;
    }
}

int main() {
    int *A, *B, *C;  // Host matrices
    int *d_A, *d_B, *d_C;  // Device matrices

    // Allocate memory for host matrices
    A = (int*)malloc(N * N * sizeof(int));
    B = (int*)malloc(N * N * sizeof(int));
    C = (int*)malloc(N * N * sizeof(int));

    // Initialize matrices A and B with random values
    for (int i = 0; i < N * N; i++) {
        A[i] = rand() % 10;
        B[i] = rand() % 10;
    }

    // Allocate memory on the device
    hipMalloc(&d_A, N * N * sizeof(int));
    hipMalloc(&d_B, N * N * sizeof(int));
    hipMalloc(&d_C, N * N * sizeof(int));

    // Copy matrices A and B from host to device
    hipMemcpy(d_A, A, N * N * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, N * N * sizeof(int), hipMemcpyHostToDevice);

    // Define the dimensions of the grid and block
    dim3 blockDim(4, 4);  // 4x4 threads per block
    dim3 gridDim((N + blockDim.x - 1) / blockDim.x, (N + blockDim.y - 1) / blockDim.y);  // Grid size

    // Launch the kernel
    matrixMulCUDA<<<gridDim, blockDim>>>(d_A, d_B, d_C, N);

    // Copy the result matrix C from device to host
    hipMemcpy(C, d_C, N * N * sizeof(int), hipMemcpyDeviceToHost);

    // Print the result matrix C
    std::cout << "Matrix C (Result of A * B):" << std::endl;
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
            std::cout << C[i * N + j] << " ";
        }
        std::cout << std::endl;
    }

    // Free memory
    free(A);
    free(B);
    free(C);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    return 0;
}
