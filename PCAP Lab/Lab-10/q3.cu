#include <iostream>
#include <hip/hip_runtime.h>

#define TILE_WIDTH 256 

__global__ void tiledConvolution1D(const float* N, const float* M, float* O, int width, int mask_width) {
    extern __shared__ float shared_mem[]; // Dynamically allocated shared memory

    int tx = threadIdx.x;
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Load elements into shared memory with padding for boundaries
    int start = blockIdx.x * blockDim.x - (mask_width / 2);
    int end = start + blockDim.x + mask_width - 1;

    // Handle boundary conditions
    if (start + tx >= 0 && start + tx < width) {
        shared_mem[tx] = N[start + tx];
    } else {
        shared_mem[tx] = 0.0f; // Padding with zeros
    }

    __syncthreads(); // Synchronize threads to ensure all data is loaded into shared memory
  
    if (idx < width) {
        float result = 0.0f;
        for (int i = 0; i < mask_width; ++i) {
            result += shared_mem[tx + i] * M[i];
        }
        O[idx] = result;
    }
}

// Host function to launch the kernel
void convolution1D(const float* h_N, const float* h_M, float* h_O, int width, int mask_width) {
    float *d_N, *d_M, *d_O;

    // Allocate device memory
    hipMalloc((void**)&d_N, width * sizeof(float));
    hipMalloc((void**)&d_M, mask_width * sizeof(float));
    hipMalloc((void**)&d_O, width * sizeof(float));

    // Copy data from host to device
    hipMemcpy(d_N, h_N, width * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_M, h_M, mask_width * sizeof(float), hipMemcpyHostToDevice);

    // Define block and grid dimensions
    int blockDim = TILE_WIDTH;
    int gridDim = (width + blockDim - 1) / blockDim;

    // Launch the kernel with dynamically allocated shared memory
    size_t shared_mem_size = (blockDim + mask_width - 1) * sizeof(float);
    tiledConvolution1D<<<gridDim, blockDim, shared_mem_size>>>(d_N, d_M, d_O, width, mask_width);

    hipMemcpy(h_O, d_O, width * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_N);
    hipFree(d_M);
    hipFree(d_O);
}

int main() {
    int width = 10; 
    int mask_width = 3; 

    float h_N[] = {1, 2, 3, 4, 5, 6, 7, 8, 9, 10};
    float h_M[] = {0.25, 0.5, 0.25};
    float h_O[width];

    convolution1D(h_N, h_M, h_O, width, mask_width);

    // Print the result
    std::cout << "Output Array: ";
    for (int i = 0; i < width; ++i) {
        std::cout << h_O[i] << " ";
    }
    std::cout << std::endl;

    return 0;
}
