#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void vectorAdd(int *A, int *B, int *C, int N, int useSingleBlock) {
    int i;
    if (useSingleBlock) {
        i = threadIdx.x;
    } else {
        i = threadIdx.x + blockIdx.x * blockDim.x;
    }
    if (i < N)
        C[i] = A[i] + B[i];
}

int main() {
    int N;
    int *A, *B, *C;
    int *d_A, *d_B, *d_C;

    printf("Enter the size of vectors (N): ");
    scanf("%d", &N);

    A = (int*) malloc(N * sizeof(int));
    B = (int*) malloc(N * sizeof(int));
    C = (int*) malloc(N * sizeof(int));

    printf("Enter %d elements for vector A: ", N);
    for (int i = 0; i < N; i++) scanf("%d", &A[i]);

    printf("Enter %d elements for vector B: ", N);
    for (int i = 0; i < N; i++) scanf("%d", &B[i]);

    hipMalloc((void**)&d_A, N * sizeof(int));
    hipMalloc((void**)&d_B, N * sizeof(int));
    hipMalloc((void**)&d_C, N * sizeof(int));

    hipMemcpy(d_A, A, N * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, N * sizeof(int), hipMemcpyHostToDevice);

    printf("\nMethod 1: Single Block, N Threads\n");
    vectorAdd<<<1, N>>>(d_A, d_B, d_C, N, 1);
    hipMemcpy(C, d_C, N * sizeof(int), hipMemcpyDeviceToHost);
    printf("Result: ");
    for (int i = 0; i < N; i++) printf("%d ", C[i]);
    printf("\n");

    printf("\nMethod 2: N Threads Total (Multiple Blocks)\n");
    int threadsPerBlock = 256;
    int blocks = (N + threadsPerBlock - 1) / threadsPerBlock;
    vectorAdd<<<blocks, threadsPerBlock>>>(d_A, d_B, d_C, N, 0);
    hipMemcpy(C, d_C, N * sizeof(int), hipMemcpyDeviceToHost);
    printf("Result: ");
    for (int i = 0; i < N; i++) printf("%d ", C[i]);
    printf("\n");

    printf("\nMethod 3: Fixed 256 Threads Per Block\n");
    vectorAdd<<<blocks, threadsPerBlock>>>(d_A, d_B, d_C, N, 0);
    hipMemcpy(C, d_C, N * sizeof(int), hipMemcpyDeviceToHost);
    printf("Result: ");
    for (int i = 0; i < N; i++) printf("%d ", C[i]);
    printf("\n");

    free(A); free(B); free(C);
    hipFree(d_A); hipFree(d_B); hipFree(d_C);

    return 0;
}
