#include <stdio.h>
#include <hip/hip_runtime.h>
#include <math.h>

__global__ void computeSine(float *input, float *output, int N) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i < N)
        output[i] = sinf(input[i]);  
}

int main() {
    int N;
    float *h_input, *h_output;
    float *d_input, *d_output;

    printf("Enter the number of elements (N): ");
    scanf("%d", &N);

    h_input = (float*) malloc(N * sizeof(float));
    h_output = (float*) malloc(N * sizeof(float));

    printf("Enter %d angles in radians: ", N);
    for (int i = 0; i < N; i++) scanf("%f", &h_input[i]);

    hipMalloc((void**)&d_input, N * sizeof(float));
    hipMalloc((void**)&d_output, N * sizeof(float));

    hipMemcpy(d_input, h_input, N * sizeof(float), hipMemcpyHostToDevice);

    int threadsPerBlock = 256;
    int blocks = (N + threadsPerBlock - 1) / threadsPerBlock;

    computeSine<<<blocks, threadsPerBlock>>>(d_input, d_output, N);

    hipMemcpy(h_output, d_output, N * sizeof(float), hipMemcpyDeviceToHost);

    printf("Sine values:\n");
    for (int i = 0; i < N; i++) printf("sin(%.4f) = %.4f\n", h_input[i], h_output[i]);

    free(h_input); free(h_output);
    hipFree(d_input); hipFree(d_output);

    return 0;
}
